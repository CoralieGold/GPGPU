#include "hip/hip_runtime.h"
/*
* TP 3 - Réduction CUDA
* --------------------------
* Mémoire paratagée, synchronisation, optimisation
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"

namespace IMAC
{
	// ==================================================== Ex 0
    __global__
    void maxReduce_ex1(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint dev_max[];

		unsigned int tid = threadIdx.x;
		unsigned int idx = tid + blockIdx.x * blockDim.x; // Id global du thread
		dev_max[idx] = dev_array[idx];
		__syncthreads();

		for(unsigned int stage = 1; stage < blockDim.x; stage *= 2)  {
			if(tid % (2*stage) == 0) {
				dev_max[tid] = max(dev_max[tid], dev_max[tid + stage]);
			}
			__syncthreads();
		}

		if(tid == 0) {
			dev_partialMax[blockIdx.x] = dev_max[0];
		}
	}

	void studentJob(const std::vector<uint> &array, const uint resCPU /* Just for comparison */)
    {
		uint *dev_array = NULL;
        const size_t bytes = array.size() * sizeof(uint);

		// Allocate array on GPU
		HANDLE_ERROR( hipMalloc( (void**)&dev_array, bytes ) );
		// Copy data from host to device
		HANDLE_ERROR( hipMemcpy( dev_array, array.data(), bytes, hipMemcpyHostToDevice ) );

		std::cout << "========== Ex 1 " << std::endl;
		uint res1 = 0; // result
		// Launch reduction and get timing
		float2 timing1 = reduce<KERNEL_EX1>(dev_array, array.size(), res1);
		
        std::cout << " -> Done: ";
        printTiming(timing1);
		compare(res1, resCPU); // Compare results

		
		std::cout << "========== Ex 2 " << std::endl;
		/// TODO

		std::cout << "========== Ex 3 " << std::endl;
		/// TODO
		
		std::cout << "========== Ex 4 " << std::endl;
		/// TODO
		
		std::cout << "========== Ex 5 " << std::endl;
		/// TODO
		

		// Free array on GPU
		hipFree( dev_array );
    }

	void printTiming(const float2 timing)
	{
		std::cout << ( timing.x < 1.f ? 1e3f * timing.x : timing.x ) << " us on device and ";
		std::cout << ( timing.y < 1.f ? 1e3f * timing.y : timing.y ) << " us on host." << std::endl;
	}

    void compare(const uint resGPU, const uint resCPU)
	{
		if (resGPU == resCPU)
		{
			std::cout << "Well done ! " << resGPU << " == " << resCPU << " !!!" << std::endl;
		}
		else
		{
			std::cout << "You failed ! " << resGPU << " != " << resCPU << " !!!" << std::endl;
		}
	}
}
