#include "hip/hip_runtime.h"
/*
* TP 3 - Réduction CUDA
* --------------------------
* Mémoire paratagée, synchronisation, optimisation
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"

namespace IMAC
{
	// ==================================================== Ex 0
    __global__
    void maxReduce_ex1(const uint *const dev_array, const uint size, uint *const dev_partialMax)
	{
		extern __shared__ uint dev_max[];
		for(int global_idx = threadIdx.x + blockIdx.x * blockDim.x;
			global_idx < size;
			global_idx += blockDim.x * gridDim.x) {

			int local_idx = threadIdx.x;
			//int global_idx = local_idx + blockIdx.x * blockDim.x; // Id global du thread
			dev_max[local_idx] = dev_array[global_idx];
			__syncthreads();

			for(unsigned int stage = blockDim.x / 2; stage > 0; stage >>= 1)  {
				if(local_idx < stage) {
					dev_max[local_idx] = umax(dev_max[local_idx], dev_max[local_idx + stage]);
				}
				__syncthreads();
			}

			if(local_idx == 0) {
				dev_partialMax[blockIdx.x] = dev_max[0];
			}
		}
		
	}

	void studentJob(const std::vector<uint> &array, const uint resCPU /* Just for comparison */)
    {
		uint *dev_array = NULL;
        const size_t bytes = array.size() * sizeof(uint);

		// Allocate array on GPU
		HANDLE_ERROR( hipMalloc( (void**)&dev_array, bytes ) );
		// Copy data from host to device
		HANDLE_ERROR( hipMemcpy( dev_array, array.data(), bytes, hipMemcpyHostToDevice ) );

		std::cout << "========== Ex 1 " << std::endl;
		uint res1 = 0; // result
		// Launch reduction and get timing
		float2 timing1 = reduce<KERNEL_EX1>(dev_array, array.size(), res1);
		
        std::cout << " -> Done: ";
        printTiming(timing1);
		compare(res1, resCPU); // Compare results

		
		std::cout << "========== Ex 2 " << std::endl;
		/// TODO

		std::cout << "========== Ex 3 " << std::endl;
		/// TODO
		
		std::cout << "========== Ex 4 " << std::endl;
		/// TODO
		
		std::cout << "========== Ex 5 " << std::endl;
		/// TODO
		

		// Free array on GPU
		hipFree( dev_array );
    }

	void printTiming(const float2 timing)
	{
		std::cout << ( timing.x < 1.f ? 1e3f * timing.x : timing.x ) << " us on device and ";
		std::cout << ( timing.y < 1.f ? 1e3f * timing.y : timing.y ) << " us on host." << std::endl;
	}

    void compare(const uint resGPU, const uint resCPU)
	{
		if (resGPU == resCPU)
		{
			std::cout << "Well done ! " << resGPU << " == " << resCPU << " !!!" << std::endl;
		}
		else
		{
			std::cout << "You failed ! " << resGPU << " != " << resCPU << " !!!" << std::endl;
		}
	}
}
