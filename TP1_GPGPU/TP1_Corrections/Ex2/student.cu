#include "hip/hip_runtime.h"
/*
* TP 1 - Premiers pas en CUDA
* --------------------------
* Ex 2: Addition de vecteurs
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{	
	// ==================================================== Kernel
	// Goal: computes the addition of two vectors (dev_a and dev_b) of size n ans stores the result in dev_res
	// We need to loop over the data if the total number of thread is smaller than the vector size
	// Given:
	// - threadIdx.x = thread id in the block
	// - blockIdx.x = block id in the grid
	// - blockDim.x = number of threads in a block
	// - gridDim.x = number of block in the grid
	// We can find:
	// -> threadIdx.x + blockIdx.x * blockDim.x = "global" id of a thread in the grid
	// -> blockDim.x * gridDim.x = total number of threads in the grid
	// So we can loop !
	// Here follow two equivalent kernels: the first use a for, the second (commented) use a while 
	__global__ void sumArraysCUDA(const int n, const int *const dev_a, const int *const dev_b, int *const dev_res)
	{
		// Loop to compute all data (if the global number of threads is smaller than array size)
		for (int 	idx = threadIdx.x + blockIdx.x * blockDim.x; // Init with the global thread id
					idx < n; // Don't exceed datat size !
					idx += blockDim.x * gridDim.x) // Add total number of threads
		{
			dev_res[idx] = dev_a[idx] + dev_b[idx];
		}
	}
	// __global__ void sumArraysCUDA(const int n, const int *const dev_a, const int *const dev_b, int *const dev_res)
	// {
	// 	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	// 	while (idx < n) 
	// 	{
	// 		dev_res[idx] = dev_a[idx] + dev_b[idx];
	// 		idx += blockDim.x * gridDim.x;
	// 	}
	// }

    void studentJob(const int size, const int *const a, const int *const b, int *const res)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		int *dev_a = NULL;
		int *dev_b = NULL;
		int *dev_res = NULL;

		// Allocate arrays on device (input and ouput)
		const size_t bytes = size * sizeof(int);
		std::cout 	<< "Allocating input (3 arrays): " 
					<< ( ( 3 * bytes ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMalloc( (void**)&dev_a, bytes ) );
		HANDLE_ERROR( hipMalloc( (void**)&dev_b, bytes ) );
		HANDLE_ERROR( hipMalloc( (void**)&dev_res, bytes ) );	
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input arrays) 
		std::cout << "Copy data from host to device" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMemcpy( dev_a, a, bytes, hipMemcpyHostToDevice ) );
		HANDLE_ERROR( hipMemcpy( dev_b, b, bytes, hipMemcpyHostToDevice ) );
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Configure number of threads/blocks
		// We should (must) verify if we don't exceed the maximum ! Use hipGetDeviceProperties ! ;-)
		const unsigned int nbThreads = 1024;
		const unsigned int nbBlocks = (size + nbThreads - 1) / nbThreads;
		
		// Launch kernel
		std::cout << "Addition on GPU (" << nbBlocks << " blocks - " << nbThreads << " threads)" << std::endl;
		chrGPU.start();
		sumArraysCUDA<<< nbBlocks, nbThreads >>>(size, dev_a, dev_b, dev_res);
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Copy data from device to host (output array)   
		std::cout << "Copy data from device to host" << std::endl;
		chrGPU.start();
		HANDLE_ERROR( hipMemcpy( res, dev_res, bytes, hipMemcpyDeviceToHost ) ); 
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl;

		// Free arrays on device
		hipFree( dev_a );
		hipFree( dev_b );
		hipFree( dev_res );
	}
}
