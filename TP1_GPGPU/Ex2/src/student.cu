#include "hip/hip_runtime.h"
/*
* TP 1 - Premiers pas en CUDA
* --------------------------
* Ex 2: Addition de vecteurs
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{
	__global__ void sumArraysCUDA(const int n, const int *const dev_a, const int *const dev_b, int *const dev_res)
	{
		int idx = threadIdx.x + blockIdx.x * blockDim.x; // Id global du thread
		const int nbThreadsGlobal = gridDim.x * blockDim.x; // Nombre de threads global

		while(idx < n)
		{
			// Somme des tableaux
			dev_res[idx] = dev_a[idx] + dev_b[idx];

			// Passage a la grid suivante
			idx += nbThreadsGlobal;
		}
	}

    void studentJob(const int size, const int *const a, const int *const b, int *const res)
	{
		// Get Cuda device properties of first device (to dynamically have max number of threads)
		hipDeviceProp_t prop;
		hipGetDeviceProperties( &prop, 0);

		ChronoGPU chrGPU;

		// 3 arrays for GPU
		int *dev_a = NULL;
		int *dev_b = NULL;
		int *dev_res = NULL;

		// Allocate arrays on device (input and ouput)
		const size_t bytes = size * sizeof(int);
		std::cout 	<< "Allocating input (3 arrays): " 
					<< ( ( 3 * bytes ) >> 20 ) << " MB on Device" << std::endl;		
		chrGPU.start();

		hipMalloc((void**) &dev_a, bytes);
		hipMalloc((void**) &dev_b, bytes);
		hipMalloc((void**) &dev_res, bytes);
		
		chrGPU.stop();
		std::cout 	<< "-> Done (allocation) : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input arrays) 
		hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, bytes, hipMemcpyHostToDevice);

		chrGPU.start();
		
		// Get maxThreadsPerBlock and number of blocks
		int nbThreads = prop.maxThreadsPerBlock;
		int nbBlocks = (size + nbThreads - 1) / nbThreads;

		std::cout << "nbBlocks = " << nbBlocks << " | nbThreads = " << nbThreads << std::endl;

		// Launch kernel
		sumArraysCUDA<<< nbBlocks, nbThreads >>>(size, dev_a, dev_b, dev_res);
		chrGPU.stop();
		std::cout 	<< "-> Done (calcul) : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from device to host (output array)  
		hipMemcpy(res, dev_res, bytes, hipMemcpyDeviceToHost);
		
		// Free arrays on device
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_res);
	}
}

